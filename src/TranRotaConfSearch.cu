#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <vector>
#include <string>
#include <math.h>
#include <time.h>
#include <stddef.h>

#include "OpenMM.h"
#include <openbabel/obconversion.h>
#include <openbabel/mol.h>
#include <hipfft/hipfft.h>
#include "ReadCrd.h"
#include "ReadGrids.h"
#include "ReadQuaternions.h"
#include "Rotate.h"
#include "GetNonbondedParameters.h"
#include "GetMinCoors.h"
#include "GetMaxCoors.h"
#include "GetIdxOfAtomsForVdwRadius.h"
#include "FillLigandGrid.h"
#include "GeneConformations.h"
#include "kernel.h"
#include "QuaternionUniformSampling.h"
#include "AddGridForcesToOpenMMSystem.h"
#include "FilterQuaternions.h"

#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 

//// main function ////
// Usage:
// TranRotaConfSearch .mol2 sys.xml grid.txt maxNumOfConf numOfRotaPerConf maxNumOfRotaPerConf numOfRotaSample nLowest mode
// maxNumOfConf: maximum num of random conformations
// numOfRotatPerConf: num of directions for each coforamtion
// mode:
// - 0: only search translation
// - 1: only search translation and rotation. The conforamtion is given in mol2 file
// - 2: search translation, rotation, and conformation

int main(int argc, char** argv)
{
  OpenMM::Platform::loadPluginsFromDirectory(
  					     "/home/xqding/apps/openmmDev/lib/plugins");

  // parse the command line parameters
  std::string mol2FileName(argv[1]);
  std::string sysFileName(argv[2]);
  std::string gridFileName(argv[3]);
  int maxNumOfConformations = atoi(argv[4]);
  int numOfRotaPerConformation = atoi(argv[5]);
  int maxNumOfRotaPerConf = atoi(argv[6]);
  int numOfRotaSample = atoi(argv[7]);
  int nLowest = atoi(argv[8]);
  int mode = atoi(argv[9]);
  if (mode == 0)
  {
    std::cout << "Model 0: only search for translation" << std::endl;
  }
  else if (mode == 1)
  {
    std::cout << "Model 1: search for translation and rotation" << std::endl;
  }
  else if (mode == 2)
  {
    std::cout << "Model 2: search for translation, rotation and conformation" << std::endl;
  }
  else
  {
    std::cout << "Error: The value of model has to be 0, 1 or 2" << std::endl;
    return 1;
  }

  // read molecule
  OpenBabel::OBMol mol;
  OpenBabel::OBConversion conv(&std::cin, &std::cout);
  conv.SetInFormat("mol2");
  conv.SetOutFormat("pdb");
  conv.ReadFile(&mol, mol2FileName);
  int nAtom = mol.NumAtoms();
  
  // read system
  std::ifstream sysFile;
  sysFile.open(sysFileName, std::ifstream::in);
  if (sysFile.fail())
  {
    std::cout << "Open system file failed: " << sysFileName << std::endl;
    return 1;
  }
  OpenMM::System *sys = new OpenMM::System();
  sys = OpenMM::XmlSerializer::deserialize<OpenMM::System>(sysFile);

  // read grid potential
  int numOfGrids, xdim, ydim, zdim;
  float midx, midy, midz;
  float xlen, ylen, zlen;
  float spacing, restraint_k;
  float *gridRadii, *gridValues;
  ReadGrids(numOfGrids, xdim, ydim, zdim,
  	    midx, midy, midz,
  	    xlen, ylen, zlen,
  	    spacing, restraint_k,
  	    gridRadii, gridValues,
  	    gridFileName);
  int numOfVdwGrids = numOfGrids - 1;

  // random clustered conformations
  double *coorsConformations;
  int numOfConformations;
  if (mode == 0 || mode == 1) // use one conformatoin in mol file
  {
    coorsConformations = new double [nAtom * 3];
    memcpy(coorsConformations, mol.GetCoordinates(), sizeof(double) * nAtom * 3);
    numOfConformations = 1;
  }
  if (mode == 2)
  {
    numOfConformations = GeneConformations(mol, sys, maxNumOfConformations, coorsConformations);
  }
  std::cout << "num of conformations: " << numOfConformations << std::endl;
  
  // get nonbonded parameters
  float atomCharges[nAtom];
  float atomEpsilons[nAtom];
  float atomRadii[nAtom];
  GetNonbondedParameters(sys, atomCharges, atomEpsilons, atomRadii);

  // get index of atoms for each vdw radius
  int numOfVdwGridsUsed;
  std::vector<int> idxOfVdwUsed;
  std::vector< std::vector<int> > idxOfAtomVdwRadius(numOfVdwGrids);
  GetIdxOfAtomsForVdwRadius(nAtom, atomRadii,
  			    numOfVdwGrids, gridRadii,
  			    numOfVdwGridsUsed, idxOfVdwUsed,
  			    idxOfAtomVdwRadius);
  int numOfGridsUsed = numOfVdwGridsUsed + 1;
  
  // copy out the potential grids which are used
  float *usedGridValues;
  usedGridValues = new float[numOfGridsUsed*xdim*ydim*zdim];
  for(int i = 0; i < numOfVdwGridsUsed; i++)
  {
    memcpy(&usedGridValues[i*xdim*ydim*zdim],
  	   &gridValues[idxOfVdwUsed[i]*xdim*ydim*zdim],
  	   sizeof(float)*xdim*ydim*zdim);
  }
  memcpy(&usedGridValues[numOfVdwGridsUsed*xdim*ydim*zdim],
  	 &gridValues[numOfVdwGrids*xdim*ydim*zdim],
  	 sizeof(float)*xdim*ydim*zdim);

  // add grid forces to OpenMM System via cumstomized forces
  double gridMinX = midx - xlen / 2;
  double gridMinY = midy - ylen / 2;
  double gridMinZ = midz - zlen / 2;
  double gridMaxX = gridMinX + (xdim - 1) * spacing;
  double gridMaxY = gridMinY + (ydim - 1) * spacing;
  double gridMaxZ = gridMinZ + (zdim - 1) * spacing;
  AddGridForcesToOpenMMSystem(xdim, ydim, zdim,
  			      gridMinX, gridMinY, gridMinZ,
  			      gridMaxX, gridMaxY, gridMaxZ,
  			      numOfVdwGridsUsed, usedGridValues,
  			      idxOfVdwUsed,
  			      idxOfAtomVdwRadius,
  			      &usedGridValues[numOfVdwGridsUsed*xdim*ydim*zdim],
  			      sys
  			      );

  // build OpenMM context
  OpenMM::LangevinIntegrator integrator(300, 10, 0.0015);
  // OpenMM::VerletIntegrator integrator(0.001);
  OpenMM::LocalEnergyMinimizer minimizer;
  
  OpenMM::Context context(*sys, integrator);
  printf( "REMARK  Add custom force Using OpenMM platform %s\n",
  	  context.getPlatform().getName().c_str() );
  OpenMM::State state;
  std::vector<OpenMM::Vec3> position(sys->getNumParticles());
  
  // batch cudaFFT for potential grids
  int n[3];
  n[0] = xdim;
  n[1] = ydim;
  n[2] = zdim;
  int inembed[3];
  inembed[0] = xdim;
  inembed[1] = ydim;
  inembed[2] = zdim;
  int idist = inembed[0] * inembed[1] * inembed[2];
  int istride = 1;
  
  int onembed[3];
  onembed[0] = xdim;
  onembed[1] = ydim;
  onembed[2] = zdim/2 + 1;
  int odist = onembed[0] * onembed[1] * onembed[2];
  int ostride = 1;
  int nBatchPotential = numOfGridsUsed;
  
  hipfftReal* d_potential_f;
  hipMalloc((void **)&d_potential_f, sizeof(hipfftReal)*nBatchPotential*idist);
  hipMemcpy(d_potential_f, usedGridValues,
  	     sizeof(hipfftReal)*nBatchPotential*idist,
  	     hipMemcpyHostToDevice);
  hipfftComplex *d_potential_F;
  hipMalloc((void **)&d_potential_F, sizeof(hipfftComplex)*nBatchPotential*odist);
  hipfftHandle potentialPlan;
  hipfftResult potentialRes = hipfftPlanMany(&potentialPlan, 3, n,
  					   inembed, istride, idist,
  					   onembed, ostride, odist,
  					   HIPFFT_R2C, nBatchPotential);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "plan creat failed!";
    return 1;
  }
  potentialRes = hipfftExecR2C(potentialPlan, d_potential_f, d_potential_F);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "transform failed!";
    return 1;
  }
  
  // generate uniform quaternions and setup data structure for all quaternions
  float* quaternions = 0;
  std::random_device rd;
  std::mt19937_64 gen(rd());
  
  // for one batch of quaternions
  int numOfQuaternionsOneBatch = 80;
  int numOfBatches = 0;

  // ligand grid for one batch
  float *ligandGridValues; // grid for ligand
  ligandGridValues = new float[numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim];
  
  // cudaFFT for ligand grid
  int nBatchLigand = numOfQuaternionsOneBatch*numOfGridsUsed;
  hipfftReal* d_ligand_f;
  hipMalloc((void **)&d_ligand_f, sizeof(hipfftReal)*nBatchLigand*idist);
  hipfftComplex * d_ligand_F;
  hipMalloc((void **)&d_ligand_F, sizeof(hipfftComplex)*nBatchLigand*odist);
  hipfftHandle ligandPlan;
  hipfftResult ligandRes = hipfftPlanMany(&ligandPlan, 3, n,
  					inembed, istride, idist,
  					onembed, ostride, odist,
  					HIPFFT_R2C, nBatchLigand);
  if (ligandRes != HIPFFT_SUCCESS)
  {
    std::cout << "plan creat failed!";
    return 1;
  }

  dim3 threads_ConjMult(1024, 1, 1);
  dim3 blocks_ConjMult((numOfQuaternionsOneBatch*numOfGridsUsed*odist)/(1024*1024) + 1,1024,1);
 
  hipfftComplex * d_ligand_sum_F;
  hipMalloc((void **)&d_ligand_sum_F, sizeof(hipfftComplex)*numOfQuaternionsOneBatch*odist);
  
  dim3 threads_SumGrids(1024, 1, 1);
  dim3 blocks_SumGrids((numOfQuaternionsOneBatch*odist)/(1024*1024) + 1,1024,1);
  
  hipfftReal *d_ligand_sum_f;
  hipMalloc((void **)&d_ligand_sum_f, sizeof(hipfftReal)*numOfQuaternionsOneBatch*idist);
  hipfftHandle ligandRPlan;
  hipfftResult ligandRRes = hipfftPlanMany(&ligandRPlan, 3, n,
  					 onembed, ostride, odist,
  					 inembed, istride, idist,
  					 HIPFFT_C2R, numOfQuaternionsOneBatch);
  if (ligandRRes != HIPFFT_SUCCESS)
  {
    std::cout << "plan creat failed!";
    return 1;
  }

  // host energy 
  float* energy;
  energy = new float[numOfQuaternionsOneBatch*idist];

  // coordinates for one conformation
  float* coor;
  coor = new float[nAtom*3];
  
  // ignore quaterions, whose end structures' dimenstion is larger than the grids
  size_t maxNQuaternionsUsed = maxNumOfConformations * numOfRotaPerConformation / numOfConformations + 1;
  if (maxNQuaternionsUsed > maxNumOfRotaPerConf)
  {
    maxNQuaternionsUsed = maxNumOfRotaPerConf;
  }
  
  size_t numOfQuaternionsUsed;
  float* quaternionsUsed = 0;  

  int *minEnergyIdxX = 0;
  int *minEnergyIdxY = 0;
  int *minEnergyIdxZ = 0;

  float *coorsUsed = 0;
  float *mincoorsUsed = 0;
  float *maxcoorsUsed = 0;
  float *ligandLengthUsed = 0;
  
  // mol for saving lowest energy pose
  OpenBabel::OBMol finalPoses[numOfConformations * nLowest];
  double energyOfFinalPoses[numOfConformations * nLowest];
  
  for(int i = 0; i < numOfConformations * nLowest; i++)
  {
    finalPoses[i] = mol;
  }
  
  // file for saving energy values of end poses 
  std::ofstream energyFile("energy.txt", std::ofstream::out);

  for (int idxOfConformer = 0; idxOfConformer < numOfConformations; idxOfConformer++)
  {
    std::cout << "idxOfConformer: " << idxOfConformer << std::endl;
    // get coordinates for one conformer
    for(int i = 0; i < nAtom; i++)
    {
      coor[i*3 + 0] = (float) coorsConformations[(idxOfConformer*nAtom + i)*3 + 0];
      coor[i*3 + 1] = (float) coorsConformations[(idxOfConformer*nAtom + i)*3 + 1];
      coor[i*3 + 2] = (float) coorsConformations[(idxOfConformer*nAtom + i)*3 + 2];
    }

    // generate all quaternions and ignore some quaternions, which will rotate the ligand
    // to have larger dimension than the grid
    if (mode == 0)
    {
      quaternions = new float[numOfRotaSample * 4]; 
      for(int i = 0; i < numOfRotaSample; i++)
      {
	quaternions[i*4 + 0] = 1;
	quaternions[i*4 + 1] = 0;
	quaternions[i*4 + 2] = 0;
	quaternions[i*4 + 3] = 0;
      }
    }
    if (mode == 1 || mode == 2)
    {
      QuaternionUniformSampling(gen, quaternions, numOfRotaSample);
    }
    
    numOfQuaternionsUsed = FilterQuaternions(coor, nAtom,
  					     numOfRotaSample, quaternions,
  					     xlen, ylen, zlen,
  					     maxNQuaternionsUsed, quaternionsUsed);

    delete[] minEnergyIdxX;
    delete[] minEnergyIdxY;
    delete[] minEnergyIdxZ;
    minEnergyIdxX = new int[numOfQuaternionsUsed];
    minEnergyIdxY = new int[numOfQuaternionsUsed];
    minEnergyIdxZ = new int[numOfQuaternionsUsed];
    std::vector <float> minEnergyQuaternionsUsed(numOfQuaternionsUsed);
    for(int i = 0; i < numOfQuaternionsUsed; i++)
    {
      minEnergyQuaternionsUsed[i] = INFINITY;
    }

    delete[] coorsUsed;
    delete[] mincoorsUsed;
    delete[] maxcoorsUsed;
    delete[] ligandLengthUsed;
    coorsUsed = new float[numOfQuaternionsUsed*nAtom*3];
    mincoorsUsed = new float[numOfQuaternionsUsed*3];
    maxcoorsUsed = new float[numOfQuaternionsUsed*3];
    ligandLengthUsed = new float[numOfQuaternionsUsed*3];

    for(int i = 0; i < numOfQuaternionsUsed; i++)
    {
      for(int j = 0; j < nAtom; j++)
      {
    	Rotate(&quaternionsUsed[i*4], &coor[j*3], &coorsUsed[i*nAtom*3+j*3]);
      }
    }
    
    // calculate minimum coor for each quaternions
    GetMinCoors(numOfQuaternionsUsed, nAtom, coorsUsed, mincoorsUsed);
    
    // calculate maximum coor for each quaternions
    GetMaxCoors(numOfQuaternionsUsed, nAtom, coorsUsed, maxcoorsUsed);

    // calculate the length for each quaternion
    for(int i = 0; i < numOfQuaternionsUsed; i++)
    {
      ligandLengthUsed[i*3 + 0] = maxcoorsUsed[i*3 + 0] - mincoorsUsed[i*3 + 0];
      ligandLengthUsed[i*3 + 1] = maxcoorsUsed[i*3 + 1] - mincoorsUsed[i*3 + 1];
      ligandLengthUsed[i*3 + 2] = maxcoorsUsed[i*3 + 2] - mincoorsUsed[i*3 + 2];
    }

    // loop over batches of quaternions
    // num of batches
    if (numOfQuaternionsUsed % numOfQuaternionsOneBatch == 0)
    {
      numOfBatches = numOfQuaternionsUsed / numOfQuaternionsOneBatch;
    }
    else
    {
      numOfBatches = numOfQuaternionsUsed / numOfQuaternionsOneBatch + 1;
    }
    
    for(int idxOfBatch = 0; idxOfBatch < numOfBatches; idxOfBatch++)
    {
      std::cout << "idxOfBatch: " << idxOfBatch << std::endl;  
      // fill ligand grid
      memset(ligandGridValues, 0, sizeof(float)*numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim);
      if ((idxOfBatch + 1) * numOfQuaternionsOneBatch > numOfQuaternionsUsed)
      {
  	FillLigandGrid(numOfQuaternionsUsed - idxOfBatch * numOfQuaternionsOneBatch,
  		       nAtom, &coorsUsed[idxOfBatch*numOfQuaternionsOneBatch*nAtom*3], &mincoorsUsed[idxOfBatch*numOfQuaternionsOneBatch*3],
  		       atomCharges, atomEpsilons,
  		       numOfVdwGridsUsed, idxOfVdwUsed,
  		       idxOfAtomVdwRadius,
  		       xdim, ydim, zdim,
  		       spacing, ligandGridValues);
      }
      else
      {	            
  	FillLigandGrid(numOfQuaternionsOneBatch,
  		       nAtom, &coorsUsed[idxOfBatch*numOfQuaternionsOneBatch*nAtom*3], &mincoorsUsed[idxOfBatch*numOfQuaternionsOneBatch*3],
  		       atomCharges, atomEpsilons,
  		       numOfVdwGridsUsed, idxOfVdwUsed,
  		       idxOfAtomVdwRadius,
  		       xdim, ydim, zdim,
  		       spacing, ligandGridValues);
      }

      // batch cudaFFT for ligand grid
      hipMemcpy(d_ligand_f, ligandGridValues,
    		 sizeof(hipfftReal)*nBatchLigand*idist,
    		 hipMemcpyHostToDevice);
      ligandRes = hipfftExecR2C(ligandPlan, d_ligand_f, d_ligand_F);
      if (ligandRes != HIPFFT_SUCCESS)
      {
    	std::cout << "ligand grid transform failed!";
    	return 1;
      }

      // calcualte energy using reverse FFT
      ConjMult <<<blocks_ConjMult, threads_ConjMult>>> (d_potential_F, d_ligand_F, odist, numOfGridsUsed);
      CUDA_CHECK();

      SumGrids <<<blocks_SumGrids, threads_SumGrids>>> (d_ligand_F, d_ligand_sum_F, numOfGridsUsed, odist, idist);
      CUDA_CHECK();

      ligandRRes = hipfftExecC2R(ligandRPlan, d_ligand_sum_F, d_ligand_sum_f);
      if (ligandRRes != HIPFFT_SUCCESS)
      {
    	std::cout << "ligand grid reverse transform failed!";
    	return 1;
      }
      
      // copy energy back
      hipMemcpy(energy, d_ligand_sum_f, sizeof(float)*numOfQuaternionsOneBatch*idist,
    		 hipMemcpyDeviceToHost);

      // record the minimum energy pose in terms of quaternions, x, y and z
      for(int q = 0; q < numOfQuaternionsOneBatch; q++)
      {
    	int idxOfQuaternions = idxOfBatch * numOfQuaternionsOneBatch + q;
  	if(idxOfQuaternions < numOfQuaternionsUsed)
  	{
  	  for(int i = 0; i < (xdim-int(ligandLengthUsed[idxOfQuaternions*3+0]/spacing)-2); i++)
  	  {
  	    for(int j = 0; j < (ydim-int(ligandLengthUsed[idxOfQuaternions*3+1]/spacing)-2); j++)
  	    {
  	      for(int k = 0; k < (ydim-int(ligandLengthUsed[idxOfQuaternions*3+2]/spacing)-2); k++)
  	      {
  		int tmp = q*idist + (i*ydim + j)*zdim + k;
    		if(energy[tmp] / sqrt(idist) < minEnergyQuaternionsUsed[idxOfQuaternions])
    		{
    		  minEnergyQuaternionsUsed[idxOfQuaternions] = energy[tmp] / sqrt(idist);
    		  minEnergyIdxX[idxOfQuaternions] = i;
    		  minEnergyIdxY[idxOfQuaternions] = j;
    		  minEnergyIdxZ[idxOfQuaternions] = k;
    		}
    	      }
    	    }
    	  }
    	}
      }      
    } // finish all batches of quaternions for one conformer
    
    // calculate the coordinates corresponding to lowest nLowest energy orientation
    std::vector<size_t> idxOfSortedQuater;
    idxOfSortedQuater = sort_index<float>(minEnergyQuaternionsUsed);
    for(int iLowest = 0; iLowest < nLowest && iLowest < numOfQuaternionsUsed; iLowest++)
    {
      int idxQ = idxOfSortedQuater[iLowest];
      double minEnergyCoorDouble[nAtom*3];
      for(int i = 0; i < nAtom; i++)
      {
    	minEnergyCoorDouble[i*3 + 0] = (double) coorsUsed[idxQ*nAtom*3 + i*3 + 0];
    	minEnergyCoorDouble[i*3 + 1] = (double) coorsUsed[idxQ*nAtom*3 + i*3 + 1];
    	minEnergyCoorDouble[i*3 + 2] = (double) coorsUsed[idxQ*nAtom*3 + i*3 + 2];
      }
    
      for(int i = 0; i < nAtom; i++)
      {
    	minEnergyCoorDouble[i*3 + 0] += (gridMinX - mincoorsUsed[idxQ*3 + 0] + minEnergyIdxX[idxQ] * spacing);
    	minEnergyCoorDouble[i*3 + 1] += (gridMinY - mincoorsUsed[idxQ*3 + 1] + minEnergyIdxY[idxQ] * spacing);
    	minEnergyCoorDouble[i*3 + 2] += (gridMinZ - mincoorsUsed[idxQ*3 + 2] + minEnergyIdxZ[idxQ] * spacing);
      }

      for(int i = 0; i < sys->getNumParticles(); i++)
      {
      	position[i] = OpenMM::Vec3(minEnergyCoorDouble[i*3+0]*OpenMM::NmPerAngstrom,
      				   minEnergyCoorDouble[i*3+1]*OpenMM::NmPerAngstrom,
      				   minEnergyCoorDouble[i*3+2]*OpenMM::NmPerAngstrom);
      }
      
      context.setPositions(position);
      if (mode == 2)
      {
	// simulated annealing
	for(int i = 700; i >= 100; i-=30)
	{
	  integrator.setTemperature(i);
	  integrator.step(500);
	}
	// minimize at the end
	minimizer.minimize(context, 0.001, 100);
      }
      // state = context.getState(OpenMM::State::Energy, false, 1<<10 | 1<<11);
      state = context.getState(OpenMM::State::Energy);
      for(int i = 0; i < sys->getNumParticles(); i++)
      {
  	minEnergyCoorDouble[i*3 + 0] = position[i][0] * OpenMM::AngstromsPerNm;
  	minEnergyCoorDouble[i*3 + 1] = position[i][1] * OpenMM::AngstromsPerNm;
  	minEnergyCoorDouble[i*3 + 2] = position[i][2] * OpenMM::AngstromsPerNm;
      }

      finalPoses[idxOfConformer * nLowest + iLowest].SetCoordinates(minEnergyCoorDouble);
      energyOfFinalPoses[idxOfConformer * nLowest + iLowest] = state.getPotentialEnergy() * OpenMM::KcalPerKJ;
      
      // mol.SetCoordinates(minEnergyCoorDouble);
      std::string fileName;
      fileName = "conformer_";
      fileName += std::to_string(idxOfConformer);
      fileName += "_";
      fileName += std::to_string(iLowest);
      fileName += ".pdb";
      // conv.WriteFile(&mol, fileName);
      conv.WriteFile(&finalPoses[idxOfConformer*nLowest+iLowest], fileName);
      energyFile << fileName << ","
      		 << idxOfConformer << ","
      		 << iLowest << ","
      		 << state.getPotentialEnergy() * OpenMM::KcalPerKJ
      		 << std::endl;
      std::cout << "Conformer: " << idxOfConformer
      		<< ", IdxQ: " << idxQ
      		<< ", IdxX: " << minEnergyIdxX[idxQ]
      		<< ", IdxY: " << minEnergyIdxY[idxQ]
      		<< ", IdxZ: " << minEnergyIdxZ[idxQ]
      		<< ", MinEnergyTranRota:" << minEnergyQuaternionsUsed[idxQ]
      		<< ", Potential Energy: " << state.getPotentialEnergy() * OpenMM::KcalPerKJ << std::endl;
    }
  }
  energyFile.close();
  return 0;
}

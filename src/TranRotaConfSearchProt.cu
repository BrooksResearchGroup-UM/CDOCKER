#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <vector>
#include <string>
#include <math.h>
#include <time.h>
#include <stddef.h>

#include "OpenMM.h"
#include <openbabel/obconversion.h>
#include <openbabel/mol.h>
#include <hipfft/hipfft.h>
#include "ReadCrd.h"
#include "ReadGrids.h"
#include "ReadQuaternions.h"
#include "Rotate.h"
#include "GetNonbondedParameters.h"
#include "GetMinCoors.h"
#include "GetMaxCoors.h"
#include "GetIdxOfAtomsForVdwRadius.h"
#include "FillLigandGrid.h"
#include "GeneConformations.h"
#include "GeneRandomConformations.h"
#include "kernel.h"
#include "QuaternionUniformSampling.h"
#include "AddCustomNonbondedForceToOpenMMSystem.h"
#include "FilterQuaternions.h"
#include "CalcRMSD.h"

#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 

//// main function ////
// Usage:
// TranRotaConfSearch ligand.mol2 ligand.xml ligand-protein.xml ligand-protein.crd grid.txt maxNumOfConf numOfRotaPerConf maxNumOfRotaPerConf numOfRotaSample nLowest mode
// Arguments:
// - ligand.mol2
// - ligand.xml: serialized xml file for ligand
// - ligand-protein.xml: serizalied xml file for both ligand and protein. The order of ligand of protein matters and the ligand should be followed by protein.
// - ligand-protein.crd: charmm crd file for both ligand and protein. The order of ligand of protein matters and the ligand should be followed by protein.
// - grid.txt: txt file for grid potential generated from protein
// - maxNumOfConf: maximum num of random conformations
// - numOfRotatPerConf: num of directions for each conformation
// - maxNumOfRotaPerConf: maximum num of direactions for each conformations
// - numOfRotaSample: num of directions random sampled, from which the valid directions are selected.
// - mode:
//   - 0: only search translation
//   - 1: only search translation and rotation. The conforamtion is given in mol2 file
//   - 2: search translation, rotation, and conformation. The final minimization step is done with presence of fixed protein


int main(int argc, char** argv)
{
  OpenMM::Platform::loadPluginsFromDirectory(
  					     "/home/xqding/apps/openmmDev/lib/plugins");

  // parse the command line parameters
  std::string mol2FileName(argv[1]);
  std::string ligandSysFileName(argv[2]);
  std::string bothSysFileName(argv[3]);
  std::string bothCrdFileName(argv[4]);
  std::string gridFileName(argv[5]);
  int maxNumOfConformations = atoi(argv[6]);
  int numOfRotaPerConformation = atoi(argv[7]);
  int maxNumOfRotaPerConf = atoi(argv[8]);
  int numOfRotaSample = atoi(argv[9]);
  int nLowest = atoi(argv[10]);
  int mode = atoi(argv[11]);

  // read ligand molecule
  OpenBabel::OBMol ligandOBMol;
  OpenBabel::OBConversion conv(&std::cin, &std::cout);
  conv.SetInFormat("mol2");
  conv.SetOutFormat("pdb");
  conv.ReadFile(&ligandOBMol, mol2FileName);
  int nAtom = ligandOBMol.NumAtoms();
  
  // read ligand openmm system
  std::ifstream ligandSysFile;
  ligandSysFile.open(ligandSysFileName, std::ifstream::in);
  if (ligandSysFile.fail())
  {
    std::cout << "Open system file failed: " << ligandSysFileName << std::endl;
    return 1;
  }
  OpenMM::System *ligandOmmSys = new OpenMM::System();
  ligandOmmSys = OpenMM::XmlSerializer::deserialize<OpenMM::System>(ligandSysFile);

  // read ligand and protein openmm system
  std::ifstream bothSysFile;
  bothSysFile.open(bothSysFileName, std::ifstream::in);
  if (bothSysFile.fail())
  {
    std::cout << "Open system file failed: " << bothSysFileName << std::endl;
    return 1;
  }
  OpenMM::System *bothOmmSys = new OpenMM::System();
  bothOmmSys = OpenMM::XmlSerializer::deserialize<OpenMM::System>(bothSysFile);

  // read coordinates for both ligand and protein
  float *bothCoor = 0;
  ReadCrd(bothOmmSys->getNumParticles(), bothCoor, bothCrdFileName);

  // read grid potential
  int numOfGrids, xdim, ydim, zdim;
  float midx, midy, midz;
  float xlen, ylen, zlen;
  float spacing, restraint_k;
  float *gridRadii, *gridValues;
  ReadGrids(numOfGrids, xdim, ydim, zdim,
  	    midx, midy, midz,
  	    xlen, ylen, zlen,
  	    spacing, restraint_k,
  	    gridRadii, gridValues,
  	    gridFileName);
  int numOfVdwGrids = numOfGrids - 1;

  // generate conformations
  double *coorsConformations;
  int numOfConformations;
  if (mode == 0 || mode == 1) // use one conformatoin in mol file
  {
    coorsConformations = new double [nAtom * 3];
    memcpy(coorsConformations, ligandOBMol.GetCoordinates(), sizeof(double) * nAtom * 3);
    numOfConformations = 1;
  }
  if (mode == 2 || mode == 3)
  {
    //numOfConformations = GeneConformations(ligandOBMol, ligandOmmSys, maxNumOfConformations, coorsConformations);
    numOfConformations = GeneRandomConformations(ligandOBMol, ligandOmmSys, maxNumOfConformations, coorsConformations);
  }
  std::cout << "num of conformations: " << numOfConformations << std::endl;
  
  // get nonbonded parameters
  float atomCharges[nAtom];
  float atomEpsilons[nAtom];
  float atomRadii[nAtom];
  GetNonbondedParameters(ligandOmmSys, atomCharges, atomEpsilons, atomRadii);

  // get index of atoms for each vdw radius
  int numOfVdwGridsUsed;
  std::vector<int> idxOfVdwUsed;
  std::vector< std::vector<int> > idxOfAtomVdwRadius(numOfVdwGrids);
  GetIdxOfAtomsForVdwRadius(nAtom, atomRadii,
  			    numOfVdwGrids, gridRadii,
  			    numOfVdwGridsUsed, idxOfVdwUsed,
  			    idxOfAtomVdwRadius);
  int numOfGridsUsed = numOfVdwGridsUsed + 1;
  
  // copy out the potential grids which are used
  float *usedGridValues;
  usedGridValues = new float[numOfGridsUsed*xdim*ydim*zdim];
  for(int i = 0; i < numOfVdwGridsUsed; i++)
  {
    memcpy(&usedGridValues[i*xdim*ydim*zdim],
  	   &gridValues[idxOfVdwUsed[i]*xdim*ydim*zdim],
  	   sizeof(float)*xdim*ydim*zdim);
  }
  memcpy(&usedGridValues[numOfVdwGridsUsed*xdim*ydim*zdim],
  	 &gridValues[numOfVdwGrids*xdim*ydim*zdim],
  	 sizeof(float)*xdim*ydim*zdim);

  // set up ligand grid context or ligand protein context
  double gridMinX = midx - xlen / 2;
  double gridMinY = midy - ylen / 2;
  double gridMinZ = midz - zlen / 2;
  double gridMaxX = gridMinX + (xdim - 1) * spacing;
  double gridMaxY = gridMinY + (ydim - 1) * spacing;
  double gridMaxZ = gridMinZ + (zdim - 1) * spacing;

  ////////////////// ligand protein ///////////////////
  // customize the nonbonded force in bothOmmSys
  AddCustomNonbondedForceToOpenMMSystem(bothOmmSys);
  // build OpenMM ligand protein context
  OpenMM::LangevinIntegrator ligandProteinIntegrator(300, 5, 0.001);
  OpenMM::LocalEnergyMinimizer ligandProteinMinimizer;
  OpenMM::Context ligandProteinContext(*bothOmmSys, ligandProteinIntegrator);
  printf( "REMARK  Build ligandProteinContext Using OpenMM platform %s\n",
	  ligandProteinContext.getPlatform().getName().c_str() );
  OpenMM::State ligandProteinState;
  std::vector<OpenMM::Vec3> ligandProteinPosition(bothOmmSys->getNumParticles());
  for(int i = 0; i < bothOmmSys->getNumParticles(); i++)
  {
    ligandProteinPosition[i] = OpenMM::Vec3(bothCoor[i*3+0]*OpenMM::NmPerAngstrom,
					    bothCoor[i*3+1]*OpenMM::NmPerAngstrom,
					    bothCoor[i*3+2]*OpenMM::NmPerAngstrom);
  }
  
  // briefly minimize the crystal structure pose of ligand and used as golden standard
  OpenBabel::OBMol ligandNativeMini = ligandOBMol;
  ligandProteinContext.setPositions(ligandProteinPosition);
  ligandProteinMinimizer.minimize(ligandProteinContext, 0.001, 1500);
  ligandProteinState = ligandProteinContext.getState(OpenMM::State::Positions);
  double tmpCoor[ligandOBMol.NumAtoms()*3];
  for(int i = 0; i < ligandOBMol.NumAtoms(); i++)
  {
    tmpCoor[i*3 + 0] = ligandProteinState.getPositions()[i][0] * OpenMM::AngstromsPerNm;
    tmpCoor[i*3 + 1] = ligandProteinState.getPositions()[i][1] * OpenMM::AngstromsPerNm;
    tmpCoor[i*3 + 2] = ligandProteinState.getPositions()[i][2] * OpenMM::AngstromsPerNm;
  }      
  ligandNativeMini.SetCoordinates(tmpCoor);
  double rmsdNativeMini = CalcRMSD(ligandOBMol, ligandNativeMini);
  
  //// cufft transform for grid potential //// 
  // batch cudaFFT for potential grids
  int n[3];
  n[0] = xdim;
  n[1] = ydim;
  n[2] = zdim;
  int inembed[3];
  inembed[0] = xdim;
  inembed[1] = ydim;
  inembed[2] = zdim;
  int idist = inembed[0] * inembed[1] * inembed[2];
  int istride = 1;
  
  int onembed[3];
  onembed[0] = xdim;
  onembed[1] = ydim;
  onembed[2] = zdim/2 + 1;
  int odist = onembed[0] * onembed[1] * onembed[2];
  int ostride = 1;
  int nBatchPotential = numOfGridsUsed;
  
  hipfftReal* d_potential_f;
  hipMalloc((void **)&d_potential_f, sizeof(hipfftReal)*nBatchPotential*idist);
  hipMemcpy(d_potential_f, usedGridValues,
  	     sizeof(hipfftReal)*nBatchPotential*idist,
  	     hipMemcpyHostToDevice);
  hipfftComplex *d_potential_F;
  hipMalloc((void **)&d_potential_F, sizeof(hipfftComplex)*nBatchPotential*odist);
  hipfftHandle potentialPlan;
  hipfftResult potentialRes = hipfftPlanMany(&potentialPlan, 3, n,
  					   inembed, istride, idist,
  					   onembed, ostride, odist,
  					   HIPFFT_R2C, nBatchPotential);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "grid potential plan creat failed!";
    return 1;
  }
  potentialRes = hipfftExecR2C(potentialPlan, d_potential_f, d_potential_F);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "grid potential cufft transform failed!";
    return 1;
  }
  ////   
  // generate uniform quaternions and setup data structure for all quaternions
  float* quaternions = 0;
  std::random_device rd;
  std::mt19937_64 gen(rd());

  //// set up one one batch of cufft transform for ligand grid ////
  // for one batch of quaternions
  int numOfQuaternionsOneBatch = 110;
  int numOfBatches = 0;

  // ligand grid for one batch
  float *ligandGridValues; // grid for ligand
  ligandGridValues = new float[numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim];
  
  // cudaFFT for ligand grid
  int nBatchLigand = numOfQuaternionsOneBatch*numOfGridsUsed;
  hipfftReal* d_ligand_f;
  hipMalloc((void **)&d_ligand_f, sizeof(hipfftReal)*nBatchLigand*idist);
  hipfftComplex * d_ligand_F;
  hipMalloc((void **)&d_ligand_F, sizeof(hipfftComplex)*nBatchLigand*odist);
  hipfftHandle ligandPlan;
  hipfftResult ligandRes = hipfftPlanMany(&ligandPlan, 3, n,
  					inembed, istride, idist,
  					onembed, ostride, odist,
  					HIPFFT_R2C, nBatchLigand);
  if (ligandRes != HIPFFT_SUCCESS)
  {
    std::cout << "ligand plan creat failed!";
    return 1;
  }

  dim3 threads_ConjMult(1024, 1, 1);
  dim3 blocks_ConjMult((numOfQuaternionsOneBatch*numOfGridsUsed*odist)/(1024*1024) + 1,1024,1);
 
  hipfftComplex * d_ligand_sum_F;
  hipMalloc((void **)&d_ligand_sum_F, sizeof(hipfftComplex)*numOfQuaternionsOneBatch*odist);
  
  dim3 threads_SumGrids(1024, 1, 1);
  dim3 blocks_SumGrids((numOfQuaternionsOneBatch*odist)/(1024*1024) + 1,1024,1);
  
  hipfftReal *d_ligand_sum_f;
  hipMalloc((void **)&d_ligand_sum_f, sizeof(hipfftReal)*numOfQuaternionsOneBatch*idist);
  hipfftHandle ligandRPlan;
  hipfftResult ligandRRes = hipfftPlanMany(&ligandRPlan, 3, n,
  					 onembed, ostride, odist,
  					 inembed, istride, idist,
  					 HIPFFT_C2R, numOfQuaternionsOneBatch);
  if (ligandRRes != HIPFFT_SUCCESS)
  {
    std::cout << "ligand reverse plan creat failed!";
    return 1;
  }
  ////
  
  // host energy 
  float* energy;
  energy = new float[numOfQuaternionsOneBatch*idist];

  // coordinates for one conformation
  float* conformerCoor;
  conformerCoor = new float[nAtom*3];
  
  // ignore quaterions, whose end structures' dimenstion is larger than the grids
  size_t maxNQuaternionsUsed = maxNumOfConformations * numOfRotaPerConformation / numOfConformations + 1;
  if (maxNQuaternionsUsed > maxNumOfRotaPerConf)
  {
    maxNQuaternionsUsed = maxNumOfRotaPerConf;
  }
  
  size_t numOfQuaternionsUsed;
  float* quaternionsUsed = 0;  

  int *minEnergyIdxX = 0;
  int *minEnergyIdxY = 0;
  int *minEnergyIdxZ = 0;

  float *coorsUsed = 0;
  float *mincoorsUsed = 0;
  float *maxcoorsUsed = 0;
  float *ligandLengthUsed = 0;
  
  // mol for saving lowest energy pose
  OpenBabel::OBMol finalPoses[numOfConformations * nLowest];
  double energyOfFinalPoses[numOfConformations * nLowest];
  
  for(int i = 0; i < numOfConformations * nLowest; i++)
  {
    finalPoses[i] = ligandOBMol;
  }
  
  // file for saving energy values of end poses 
  std::ofstream energyFile("energy.txt", std::ofstream::out);

  ////////////////////////
  //// start searching ///
  ////////////////////////
  for (int idxOfConformer = 0; idxOfConformer < numOfConformations; idxOfConformer++)
  {
    std::cout << "idxOfConformer: " << idxOfConformer << std::endl;
    // get coordinates for one conformer
    for(int i = 0; i < nAtom; i++)
    {
      conformerCoor[i*3 + 0] = (float) coorsConformations[(idxOfConformer*nAtom + i)*3 + 0];
      conformerCoor[i*3 + 1] = (float) coorsConformations[(idxOfConformer*nAtom + i)*3 + 1];
      conformerCoor[i*3 + 2] = (float) coorsConformations[(idxOfConformer*nAtom + i)*3 + 2];
    }

    // generate all quaternions and ignore some quaternions, which will rotate the ligand
    // to have larger dimension than the grid
    if (mode == 0) // one search traslation, so make all quaterions unity.
    {
      quaternions = new float[numOfRotaSample * 4]; 
      for(int i = 0; i < numOfRotaSample; i++)
      {
	quaternions[i*4 + 0] = 1;
	quaternions[i*4 + 1] = 0;
	quaternions[i*4 + 2] = 0;
	quaternions[i*4 + 3] = 0;
      }
    }
    if (mode == 1 || mode == 2)
    {
      QuaternionUniformSampling(gen, quaternions, numOfRotaSample);
    }
    
    numOfQuaternionsUsed = FilterQuaternions(conformerCoor, nAtom,
  					     numOfRotaSample, quaternions,
  					     xlen, ylen, zlen,
  					     maxNQuaternionsUsed, quaternionsUsed);

    delete[] minEnergyIdxX;
    delete[] minEnergyIdxY;
    delete[] minEnergyIdxZ;
    minEnergyIdxX = new int[numOfQuaternionsUsed];
    minEnergyIdxY = new int[numOfQuaternionsUsed];
    minEnergyIdxZ = new int[numOfQuaternionsUsed];
    std::vector <float> minEnergyQuaternionsUsed(numOfQuaternionsUsed);
    for(int i = 0; i < numOfQuaternionsUsed; i++)
    {
      minEnergyQuaternionsUsed[i] = INFINITY;
    }

    delete[] coorsUsed;
    delete[] mincoorsUsed;
    delete[] maxcoorsUsed;
    delete[] ligandLengthUsed;
    coorsUsed = new float[numOfQuaternionsUsed*nAtom*3];
    mincoorsUsed = new float[numOfQuaternionsUsed*3];
    maxcoorsUsed = new float[numOfQuaternionsUsed*3];
    ligandLengthUsed = new float[numOfQuaternionsUsed*3];

    for(int i = 0; i < numOfQuaternionsUsed; i++)
    {
      for(int j = 0; j < nAtom; j++)
      {
    	Rotate(&quaternionsUsed[i*4], &conformerCoor[j*3], &coorsUsed[i*nAtom*3+j*3]);
      }
    }
    
    // calculate minimum coor for each quaternions
    GetMinCoors(numOfQuaternionsUsed, nAtom, coorsUsed, mincoorsUsed);
    
    // calculate maximum coor for each quaternions
    GetMaxCoors(numOfQuaternionsUsed, nAtom, coorsUsed, maxcoorsUsed);

    // calculate the length for each quaternion
    for(int i = 0; i < numOfQuaternionsUsed; i++)
    {
      ligandLengthUsed[i*3 + 0] = maxcoorsUsed[i*3 + 0] - mincoorsUsed[i*3 + 0];
      ligandLengthUsed[i*3 + 1] = maxcoorsUsed[i*3 + 1] - mincoorsUsed[i*3 + 1];
      ligandLengthUsed[i*3 + 2] = maxcoorsUsed[i*3 + 2] - mincoorsUsed[i*3 + 2];
    }

    // loop over batches of quaternions
    // num of batches
    if (numOfQuaternionsUsed % numOfQuaternionsOneBatch == 0)
    {
      numOfBatches = numOfQuaternionsUsed / numOfQuaternionsOneBatch;
    }
    else
    {
      numOfBatches = numOfQuaternionsUsed / numOfQuaternionsOneBatch + 1;
    }
    
    for(int idxOfBatch = 0; idxOfBatch < numOfBatches; idxOfBatch++)
    {
      std::cout << "idxOfBatch: " << idxOfBatch << std::endl;  
      // fill ligand grid
      memset(ligandGridValues, 0, sizeof(float)*numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim);
      if ((idxOfBatch + 1) * numOfQuaternionsOneBatch > numOfQuaternionsUsed)
      {
  	FillLigandGrid(numOfQuaternionsUsed - idxOfBatch * numOfQuaternionsOneBatch,
  		       nAtom, &coorsUsed[idxOfBatch*numOfQuaternionsOneBatch*nAtom*3], &mincoorsUsed[idxOfBatch*numOfQuaternionsOneBatch*3],
  		       atomCharges, atomEpsilons,
  		       numOfVdwGridsUsed, idxOfVdwUsed,
  		       idxOfAtomVdwRadius,
  		       xdim, ydim, zdim,
  		       spacing, ligandGridValues);
      }
      else
      {	            
  	FillLigandGrid(numOfQuaternionsOneBatch,
  		       nAtom, &coorsUsed[idxOfBatch*numOfQuaternionsOneBatch*nAtom*3], &mincoorsUsed[idxOfBatch*numOfQuaternionsOneBatch*3],
  		       atomCharges, atomEpsilons,
  		       numOfVdwGridsUsed, idxOfVdwUsed,
  		       idxOfAtomVdwRadius,
  		       xdim, ydim, zdim,
  		       spacing, ligandGridValues);
      }

      // batch cudaFFT for ligand grid
      hipMemcpy(d_ligand_f, ligandGridValues,
    		 sizeof(hipfftReal)*nBatchLigand*idist,
    		 hipMemcpyHostToDevice);
      ligandRes = hipfftExecR2C(ligandPlan, d_ligand_f, d_ligand_F);
      if (ligandRes != HIPFFT_SUCCESS)
      {
    	std::cout << "ligand grid transform failed!";
    	return 1;
      }

      // calcualte energy using reverse FFT
      ConjMult <<<blocks_ConjMult, threads_ConjMult>>> (d_potential_F, d_ligand_F, odist, numOfGridsUsed);
      CUDA_CHECK();

      SumGrids <<<blocks_SumGrids, threads_SumGrids>>> (d_ligand_F, d_ligand_sum_F, numOfGridsUsed, odist, idist);
      CUDA_CHECK();

      ligandRRes = hipfftExecC2R(ligandRPlan, d_ligand_sum_F, d_ligand_sum_f);
      if (ligandRRes != HIPFFT_SUCCESS)
      {
    	std::cout << "ligand grid reverse transform failed!";
    	return 1;
      }
      
      // copy energy back
      hipMemcpy(energy, d_ligand_sum_f, sizeof(float)*numOfQuaternionsOneBatch*idist,
    		 hipMemcpyDeviceToHost);

      // record the minimum energy pose in terms of quaternions, x, y and z
      for(int q = 0; q < numOfQuaternionsOneBatch; q++)
      {
    	int idxOfQuaternions = idxOfBatch * numOfQuaternionsOneBatch + q;
  	if(idxOfQuaternions < numOfQuaternionsUsed)
  	{
  	  for(int i = 0; i < (xdim-int(ligandLengthUsed[idxOfQuaternions*3+0]/spacing)-2); i++)
  	  {
  	    for(int j = 0; j < (ydim-int(ligandLengthUsed[idxOfQuaternions*3+1]/spacing)-2); j++)
  	    {
  	      for(int k = 0; k < (ydim-int(ligandLengthUsed[idxOfQuaternions*3+2]/spacing)-2); k++)
  	      {
  		int tmp = q*idist + (i*ydim + j)*zdim + k;
    		if(energy[tmp] / sqrt(idist) < minEnergyQuaternionsUsed[idxOfQuaternions])
    		{
    		  minEnergyQuaternionsUsed[idxOfQuaternions] = energy[tmp] / sqrt(idist);
    		  minEnergyIdxX[idxOfQuaternions] = i;
    		  minEnergyIdxY[idxOfQuaternions] = j;
    		  minEnergyIdxZ[idxOfQuaternions] = k;
    		}
    	      }
    	    }
    	  }
    	}
      }      
    } // finish all batches of quaternions for one conformer
    
    // calculate the coordinates corresponding to lowest nLowest energy orientation
    std::vector<size_t> idxOfSortedQuater;
    idxOfSortedQuater = sort_index<float>(minEnergyQuaternionsUsed);
    for(int iLowest = 0; iLowest < nLowest && iLowest < numOfQuaternionsUsed; iLowest++)
    {
      int idxQ = idxOfSortedQuater[iLowest];
      double minEnergyCoorDouble[nAtom*3];
      
      // coordinate corresponding to the lowest energy pose in term of orientations
      for(int i = 0; i < nAtom; i++)
      {
    	minEnergyCoorDouble[i*3 + 0] = (double) coorsUsed[idxQ*nAtom*3 + i*3 + 0];
    	minEnergyCoorDouble[i*3 + 1] = (double) coorsUsed[idxQ*nAtom*3 + i*3 + 1];
    	minEnergyCoorDouble[i*3 + 2] = (double) coorsUsed[idxQ*nAtom*3 + i*3 + 2];
      }      
      for(int i = 0; i < nAtom; i++)
      {
    	minEnergyCoorDouble[i*3 + 0] += (gridMinX - mincoorsUsed[idxQ*3 + 0] + minEnergyIdxX[idxQ] * spacing);
    	minEnergyCoorDouble[i*3 + 1] += (gridMinY - mincoorsUsed[idxQ*3 + 1] + minEnergyIdxY[idxQ] * spacing);
    	minEnergyCoorDouble[i*3 + 2] += (gridMinZ - mincoorsUsed[idxQ*3 + 2] + minEnergyIdxZ[idxQ] * spacing);
      }

      // final minimize
      // minimize with presence of fixed protein
      if (mode == 2)
      {
	for(int i = 0; i < ligandOmmSys->getNumParticles(); i++) // only update position of ligand
	{
	  ligandProteinPosition[i] = OpenMM::Vec3(minEnergyCoorDouble[i*3+0]*OpenMM::NmPerAngstrom,
						  minEnergyCoorDouble[i*3+1]*OpenMM::NmPerAngstrom,
						  minEnergyCoorDouble[i*3+2]*OpenMM::NmPerAngstrom);
	}
	ligandProteinContext.setPositions(ligandProteinPosition);
	ligandProteinMinimizer.minimize(ligandProteinContext, 0.01, 1000);
	ligandProteinState = ligandProteinContext.getState(OpenMM::State::Energy|OpenMM::State::Positions);
	for(int i = 0; i < ligandOmmSys->getNumParticles(); i++)
	{
	  minEnergyCoorDouble[i*3 + 0] = ligandProteinState.getPositions()[i][0] * OpenMM::AngstromsPerNm;
	  minEnergyCoorDouble[i*3 + 1] = ligandProteinState.getPositions()[i][1] * OpenMM::AngstromsPerNm;
	  minEnergyCoorDouble[i*3 + 2] = ligandProteinState.getPositions()[i][2] * OpenMM::AngstromsPerNm;
	}      
	energyOfFinalPoses[idxOfConformer * nLowest + iLowest] = ligandProteinState.getPotentialEnergy() * OpenMM::KcalPerKJ;
      }
      
      // write nlowest energy pose out
      finalPoses[idxOfConformer * nLowest + iLowest].SetCoordinates(minEnergyCoorDouble);
      double rmsd0 = CalcRMSD(finalPoses[idxOfConformer * nLowest + iLowest], ligandOBMol);
      double rmsd1 = CalcRMSD(finalPoses[idxOfConformer * nLowest + iLowest], ligandNativeMini);
      
      std::string fileName;
      fileName = "conformer_";
      fileName += std::to_string(idxOfConformer);
      fileName += "_";
      fileName += std::to_string(iLowest);
      fileName += ".pdb";
      // conv.WriteFile(&finalPoses[idxOfConformer*nLowest+iLowest], fileName);
      energyFile << fileName << ","
      		 << idxOfConformer << ","
      		 << iLowest << ","
      		 << energyOfFinalPoses[idxOfConformer * nLowest + iLowest] << ","
		 << rmsdNativeMini << ","
		 << rmsd0 << ","
		 << rmsd1 << ","
      		 << std::endl;
      std::cout << "Conformer: " << idxOfConformer
      		<< ", IdxQ: " << idxQ
      		<< ", IdxX: " << minEnergyIdxX[idxQ]
      		<< ", IdxY: " << minEnergyIdxY[idxQ]
      		<< ", IdxZ: " << minEnergyIdxZ[idxQ]
      		<< ", MinEnergyTranRota:" << minEnergyQuaternionsUsed[idxQ]
      		<< ", Potential Energy: " << energyOfFinalPoses[idxOfConformer * nLowest + iLowest]
		<< ", RMSDNativeMini:" << rmsdNativeMini
		<< ", RMSD0: " << rmsd0
		<< ", RMSD1: " << rmsd1
		<< std::endl;
    }
  }  
  energyFile.close();
  return 0;
}

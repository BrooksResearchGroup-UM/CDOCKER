#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <vector>
#include <string>
#include <math.h>
#include <time.h>

#include "OpenMM.h"
#include <openbabel/obconversion.h>
#include <openbabel/mol.h>
#include <hipfft/hipfft.h>
#include "ReadCrd.h"
#include "ReadGrids.h"
#include "ReadQuaternions.h"
#include "Rotate.h"
#include "GetNonbondedParameters.h"
#include "GetMinCoors.h"
#include "GetMaxCoors.h"
#include "GetIdxOfAtomsForVdwRadius.h"
#include "FillLigandGrid.h"
#include "GeneConformations.h"
#include "kernel.h"

#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 

//// main function ////
int main(int argc, char** argv)
{
  OpenMM::Platform::loadPluginsFromDirectory(
  					     "/home/xqding/apps/openmmDev/lib/plugins");
  
  // read molecule
  std::string fileName(argv[1]);
  OpenBabel::OBMol mol;
  OpenBabel::OBConversion conv(&std::cin, &std::cout);
  conv.SetInFormat("mol2");
  conv.SetOutFormat("pdb");
  conv.ReadFile(&mol, fileName);
  int nAtom = mol.NumAtoms();
  
  // read system
  std::ifstream sysFile;
  sysFile.open(argv[2], std::ifstream::in);
  if (sysFile.fail())
  {
    std::cout << "Open system file failed: " << argv[4] << std::endl;    return 1;
  }
  OpenMM::System *sys = new OpenMM::System();
  sys = OpenMM::XmlSerializer::deserialize<OpenMM::System>(sysFile);

  // // random clustered conformations
  // double *coorsConformations;
  // int numOfConformations;
  // numOfConformations = GeneConformations(mol, sys, coorsConformations);
  // std::cout << "numOfConformations: " << numOfConformations << std::endl;
  
  // get nonbonded parameters
  float atomCharges[nAtom];
  float atomEpsilons[nAtom];
  float atomRadii[nAtom];
  GetNonbondedParameters(sys, atomCharges, atomEpsilons, atomRadii);

  // read grid potential
  int numOfGrids, xdim, ydim, zdim;
  float midx, midy, midz;
  float xlen, ylen, zlen;
  float spacing, restraint_k;
  float *gridRadii, *gridValues;
  ReadGrids(numOfGrids, xdim, ydim, zdim,
  	    midx, midy, midz,
  	    xlen, ylen, zlen,
  	    spacing, restraint_k,
  	    gridRadii, gridValues,
  	    argv[3]);
  int numOfVdwGrids = numOfGrids - 1;
  
  // add electrostatic grid force
  double xmin = midx - xlen / 2;
  double ymin = midy - ylen / 2;
  double zmin = midz - zlen / 2;
  double xmax = xmin + (xdim - 1) * spacing;
  double ymax = ymin + (ydim - 1) * spacing;
  double zmax = zmin + (zdim - 1) * spacing;

  // std::vector <double> tmpGridValue(xdim*ydim*zdim, 0);
  // for(int i = 0; i < xdim*ydim*zdim; i++)
  // {
  //   tmpGridValue[i] = gridValues[numOfVdwGrids*xdim*ydim*zdim + i] * OpenMM::KJPerKcal;
  // }
  // OpenMM::Continuous3DFunction *elecGridFunction =
  //   new OpenMM::Continuous3DFunction(xdim, ydim, zdim,
  // 				     tmpGridValue,
  // 				     xmin*OpenMM::NmPerAngstrom,
  // 				     xmax*OpenMM::NmPerAngstrom,
  // 				     ymin*OpenMM::NmPerAngstrom,
  // 				     ymax*OpenMM::NmPerAngstrom,
  // 				     zmin*OpenMM::NmPerAngstrom,
  // 				     zmax*OpenMM::NmPerAngstrom);
  
  // OpenMM::CustomCompoundBondForce *elecGridPotential =
  //   new OpenMM::CustomCompoundBondForce(1, "elecGrid(x1,y1,z1) * q");
  // elecGridPotential->setForceGroup(10);
  // int idxElecGrid = sys->addForce(elecGridPotential);
  // elecGridPotential->addTabulatedFunction("elecGrid", elecGridFunction);
  // elecGridPotential->addPerBondParameter("q");
  
  // std::vector<int> idxParticle(1,0);
  // std::vector<double> parameter(1,0);
  
  // for (int i = 0; i < sys->getNumParticles(); i++)
  // {
  //   idxParticle[0] = i;
  //   parameter[0] = atomCharges[i];
  //   elecGridPotential->addBond(idxParticle, parameter);
  // }

  // get index of atoms for each vdw radius
  int numOfVdwGridsUsed;
  std::vector<int> idxOfVdwUsed;
  std::vector< std::vector<int> > idxOfAtomVdwRadius(numOfVdwGrids);
  GetIdxOfAtomsForVdwRadius(nAtom, atomRadii,
  			    numOfVdwGrids, gridRadii,
  			    numOfVdwGridsUsed, idxOfVdwUsed,
  			    idxOfAtomVdwRadius);
  int numOfGridsUsed = numOfVdwGridsUsed + 1;

  // copy out the potential grids which are used
  float *usedGridValues;
  usedGridValues = new float[numOfGridsUsed*xdim*ydim*zdim];
  for(int i = 0; i < numOfVdwGridsUsed; i++)
  {
    memcpy(&usedGridValues[i*xdim*ydim*zdim],
  	   &gridValues[idxOfVdwUsed[i]*xdim*ydim*zdim],
  	   sizeof(float)*xdim*ydim*zdim);
  }
  memcpy(&usedGridValues[numOfVdwGridsUsed*xdim*ydim*zdim],
  	 &gridValues[numOfVdwGrids*xdim*ydim*zdim],
  	 sizeof(float)*xdim*ydim*zdim);

  // // add vdw grid force
  // OpenMM::Continuous3DFunction *vdwGridFunctions[numOfVdwGridsUsed];
  // OpenMM::CustomCompoundBondForce *vdwGridPotentials[numOfVdwGridsUsed];

  // std::string formula;
  // for(int k = 0; k < numOfVdwGridsUsed; k++)
  // {
  //   for(int i = 0; i < xdim*ydim*zdim; i++)
  //   {
  //     tmpGridValue[i] = usedGridValues[k*xdim*ydim*zdim + i] * OpenMM::KJPerKcal;
  //   }
  //   vdwGridFunctions[k] =
  //     new OpenMM::Continuous3DFunction(xdim, ydim, zdim,
  // 				       tmpGridValue,
  // 				       xmin * OpenMM::NmPerAngstrom,
  // 				       xmax * OpenMM::NmPerAngstrom,
  // 				       ymin * OpenMM::NmPerAngstrom,
  // 				       ymax * OpenMM::NmPerAngstrom,
  // 				       zmin * OpenMM::NmPerAngstrom,
  // 				       zmax * OpenMM::NmPerAngstrom);
  //   formula = "vdwGrid";
  //   formula += std::to_string(k);
  //   formula += "(x1,y1,z1) * sqrt(epsilon)";
    
  //   vdwGridPotentials[k] = new OpenMM::CustomCompoundBondForce(1, formula);
  //   vdwGridPotentials[k]->setForceGroup(11);
  //   sys->addForce(vdwGridPotentials[k]);
  //   formula = "vdwGrid";
  //   formula += std::to_string(k);
	
  //   vdwGridPotentials[k]->addTabulatedFunction(formula, vdwGridFunctions[k]);
  //   vdwGridPotentials[k]->addPerBondParameter("epsilon");

  //   for (int i = 0; i < idxOfAtomVdwRadius[idxOfVdwUsed[k]].size(); i++)
  //   {
  //     int idx = idxOfAtomVdwRadius[idxOfVdwUsed[k]][i];
  //     idxParticle[0] = idx;
  //     parameter[0] = atomEpsilons[idx];
  //     vdwGridPotentials[k]->addBond(idxParticle, parameter);
  //   }
  // }
  
  // // get the energy
  // OpenMM::VerletIntegrator integrator(0.001);
  // OpenMM::Context context(*sys, integrator);
  // printf( "REMARK  Add custom force Using OpenMM platform %s\n",
  // 	  context.getPlatform().getName().c_str() );

  // OpenMM::State state;
  // std::vector<OpenMM::Vec3> position(sys->getNumParticles());

  // for(int i = 0; i < sys->getNumParticles(); i++)
  // {
  //   position[i] = OpenMM::Vec3(mol.GetCoordinates()[i*3+0]*OpenMM::NmPerAngstrom,
  // 			       mol.GetCoordinates()[i*3+1]*OpenMM::NmPerAngstrom,
  // 			       mol.GetCoordinates()[i*3+2]*OpenMM::NmPerAngstrom);
  // }
  // context.setPositions(position);

  // state = context.getState(OpenMM::State::Energy);
  // std::cout << "Potential Energy: " << state.getPotentialEnergy() * OpenMM::KcalPerKJ << std::endl;

  //// do translation and rotation search using FFT
  // batch cudaFFT for potential grids
  int n[3];
  n[0] = xdim;
  n[1] = ydim;
  n[2] = zdim;
  int inembed[3];
  inembed[0] = xdim;
  inembed[1] = ydim;
  inembed[2] = zdim;
  int idist = inembed[0] * inembed[1] * inembed[2];
  int istride = 1;
  
  int onembed[3];
  onembed[0] = xdim;
  onembed[1] = ydim;
  onembed[2] = zdim/2 + 1;
  int odist = onembed[0] * onembed[1] * onembed[2];
  int ostride = 1;
  int nBatchPotential = numOfGridsUsed;
  
  hipfftReal* d_potential_f;
  hipMalloc((void **)&d_potential_f, sizeof(hipfftReal)*nBatchPotential*idist);
  hipMemcpy(d_potential_f, usedGridValues,
  	     sizeof(hipfftReal)*nBatchPotential*idist,
  	     hipMemcpyHostToDevice);
  hipfftComplex *d_potential_F;
  hipMalloc((void **)&d_potential_F, sizeof(hipfftComplex)*nBatchPotential*odist);
  hipfftHandle potentialPlan;
  hipfftResult potentialRes = hipfftPlanMany(&potentialPlan, 3, n,
  					   inembed, istride, idist,
  					   onembed, ostride, odist,
  					   HIPFFT_R2C, nBatchPotential);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "plan creat failed!";
    return 1;
  }
  potentialRes = hipfftExecR2C(potentialPlan, d_potential_f, d_potential_F);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "transform failed!";
    return 1;
  }

  // read quaternioins
  int numOfTotalQuaternions = atoi(argv[4]);
  float *quaternioins;
  quaternioins = new float[numOfTotalQuaternions * 4];
  ReadQuaternions(numOfTotalQuaternions, quaternioins, argv[5]);
  
  // loop over all batches for different orientation
  int numOfQuaternionsOneBatch = 100;
  int numOfBatches = numOfTotalQuaternions / numOfQuaternionsOneBatch + 1;
    
  // allocate the data structures which will be used
  float *coors; // rotated coordinates
  coors = new float[numOfQuaternionsOneBatch*nAtom*3];
  float mincoors[numOfQuaternionsOneBatch*3]; // minimium coordinates along x, y, and z
  float maxcoors[numOfQuaternionsOneBatch*3]; // maximum coordinates along x, y, and z
  float ligandLength[numOfQuaternionsOneBatch*3]; // lenth along x, y and z for each orientation
  
  float *ligandGridValues; // grid for ligand
  ligandGridValues = new float[numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim];
  
  // cudaFFT for ligand grid
  int nBatchLigand = numOfQuaternionsOneBatch*numOfGridsUsed;
  hipfftReal* d_ligand_f;
  hipMalloc((void **)&d_ligand_f, sizeof(hipfftReal)*nBatchLigand*idist);
  hipfftComplex * d_ligand_F;
  hipMalloc((void **)&d_ligand_F, sizeof(hipfftComplex)*nBatchLigand*odist);
  hipfftHandle ligandPlan;
  hipfftResult ligandRes = hipfftPlanMany(&ligandPlan, 3, n,
  					inembed, istride, idist,
  					onembed, ostride, odist,
  					HIPFFT_R2C, nBatchLigand);
  if (ligandRes != HIPFFT_SUCCESS)
  {
    std::cout << "plan creat failed!";
    return 1;
  }

  dim3 threads_ConjMult(1024, 1, 1);
  dim3 blocks_ConjMult((numOfQuaternionsOneBatch*numOfGridsUsed*odist)/(1024*1024) + 1,1024,1);
 
  hipfftComplex * d_ligand_sum_F;
  hipMalloc((void **)&d_ligand_sum_F, sizeof(hipfftComplex)*numOfQuaternionsOneBatch*odist);
  
  dim3 threads_SumGrids(1024, 1, 1);
  dim3 blocks_SumGrids((numOfQuaternionsOneBatch*odist)/(1024*1024) + 1,1024,1);
  
  hipfftReal *d_ligand_sum_f;
  hipMalloc((void **)&d_ligand_sum_f, sizeof(hipfftReal)*numOfQuaternionsOneBatch*idist);
  hipfftHandle ligandRPlan;
  hipfftResult ligandRRes = hipfftPlanMany(&ligandRPlan, 3, n,
  					 onembed, ostride, odist,
  					 inembed, istride, idist,
  					 HIPFFT_C2R, numOfQuaternionsOneBatch);
  if (ligandRRes != HIPFFT_SUCCESS)
  {
    std::cout << "plan creat failed!";
    return 1;
  }
  float* energy;
  energy = new float[numOfQuaternionsOneBatch*idist];

  // get coor
  float* coor;
  coor = new float[nAtom*3];
  for(int i = 0; i < nAtom; i++)
  {
    coor[i*3 + 0] = (float) mol.GetCoordinates()[i*3 + 0];
    coor[i*3 + 1] = (float) mol.GetCoordinates()[i*3 + 1];
    coor[i*3 + 2] = (float) mol.GetCoordinates()[i*3 + 2];
  }

  int minEnergyQ = 0;
  int minEnergyIdxX = 0;
  int minEnergyIdxY = 0;
  int minEnergyIdxZ = 0;
  float minEnergy = INFINITY;
    
  for(int idxOfBatch = 0; idxOfBatch < numOfBatches; idxOfBatch++)
  {
    std::cout << "idxOfBatch: " << idxOfBatch << std::endl;
    // rotate
    for(int i = 0; i < numOfQuaternionsOneBatch; i++)
    {
      if (idxOfBatch*numOfQuaternionsOneBatch + i < numOfTotalQuaternions)
      {
	for(int j = 0; j < nAtom; j++)
	{
	  Rotate(&quaternioins[(idxOfBatch*numOfQuaternionsOneBatch + i)*4], &coor[j*3], &coors[i*nAtom*3+j*3]);
	}
      }
    }
    
    // calculate minimum coor for each quaternions
    GetMinCoors(numOfQuaternionsOneBatch, nAtom, coors, mincoors);

    // calculate maximum coor for each quaternions
    GetMaxCoors(numOfQuaternionsOneBatch, nAtom, coors, maxcoors);

    // calculate the length for each quaternion
    for(int i = 0; i < numOfQuaternionsOneBatch; i++)
    {
      ligandLength[i*3 + 0] = maxcoors[i*3 + 0] - mincoors[i*3 + 0];
      ligandLength[i*3 + 1] = maxcoors[i*3 + 1] - mincoors[i*3 + 1];
      ligandLength[i*3 + 2] = maxcoors[i*3 + 2] - mincoors[i*3 + 2];
    }
      
    // fill ligand grid
    memset(ligandGridValues, 0, sizeof(float)*numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim);
    FillLigandGrid(numOfQuaternionsOneBatch,
		   nAtom, coors, mincoors,
		   atomCharges, atomEpsilons,
		   numOfVdwGridsUsed, idxOfVdwUsed,
		   idxOfAtomVdwRadius,
		   xdim, ydim, zdim,
		   spacing, ligandGridValues);


    // batch cudaFFT for ligand grid
    hipMemcpy(d_ligand_f, ligandGridValues,
	       sizeof(hipfftReal)*nBatchLigand*idist,
	       hipMemcpyHostToDevice);
    ligandRes = hipfftExecR2C(ligandPlan, d_ligand_f, d_ligand_F);
    if (ligandRes != HIPFFT_SUCCESS)
    {
      std::cout << "transform failed!";
      return 1;
    }

    // calcualte energy using reverse FFT
    ConjMult <<<blocks_ConjMult, threads_ConjMult>>> (d_potential_F, d_ligand_F, odist, numOfGridsUsed);
    CUDA_CHECK();

    SumGrids <<<blocks_SumGrids, threads_SumGrids>>> (d_ligand_F, d_ligand_sum_F, numOfGridsUsed, odist, idist);
    CUDA_CHECK();

    ligandRRes = hipfftExecC2R(ligandRPlan, d_ligand_sum_F, d_ligand_sum_f);
    if (ligandRRes != HIPFFT_SUCCESS)
    {
      std::cout << "transform failed!";
      return 1;
    }

    // copy energy back
    hipMemcpy(energy, d_ligand_sum_f, sizeof(float)*numOfQuaternionsOneBatch*idist,
	       hipMemcpyDeviceToHost);

    // record the minimum energy pose in terms of quaternions, x, y and z
    for(int q = 0; q < numOfQuaternionsOneBatch; q++)
    {
      for(int i = 0; i < (xdim-int(ligandLength[q*3+0]/spacing)-2); i++)
      {
	for(int j = 0; j < (ydim-int(ligandLength[q*3+1]/spacing)-2); j++)
	{
	  for(int k = 0; k < (zdim-int(ligandLength[q*3+2]/spacing)-2); k++)
	  {
	    if(idxOfBatch*numOfQuaternionsOneBatch + q < numOfTotalQuaternions)
	    {
	      int tmp = q*idist + (i*ydim + j)*zdim + k;
	      if((energy[tmp]/sqrt(idist)) < minEnergy)
	      {
		minEnergy = energy[tmp] / sqrt(idist);
		minEnergyQ = idxOfBatch * numOfQuaternionsOneBatch + q;
		minEnergyIdxX = i;
		minEnergyIdxY = j;
		minEnergyIdxZ = k;
	      }
	    }
	  }
	}
      }
    }
  }

  // calculate the coordinates corresponding to minimum energy
  float minEnergyCoor[nAtom*3];
  for(int i = 0; i < nAtom; i++)
  {
    Rotate(&quaternioins[minEnergyQ*4], &coor[i*3], &minEnergyCoor[i*3]);
  }
  
  float minEnergyMinX = minEnergyCoor[0];
  float minEnergyMinY = minEnergyCoor[1];
  float minEnergyMinZ = minEnergyCoor[2];
  for(int i = 1; i < nAtom; i++)
  {
    if (minEnergyCoor[i*3+0] < minEnergyMinX) { minEnergyMinX = minEnergyCoor[i*3+0]; }
    if (minEnergyCoor[i*3+1] < minEnergyMinY) { minEnergyMinY = minEnergyCoor[i*3+1]; }
    if (minEnergyCoor[i*3+2] < minEnergyMinZ) { minEnergyMinZ = minEnergyCoor[i*3+2]; }
  }

  // std::cout << "minEnergyMinX: " << minEnergyMinX << std::endl;
  // std::cout << "minEnergyMinY: " << minEnergyMinY << std::endl;
  // std::cout << "minEnergyMinZ: " << minEnergyMinZ << std::endl;
  
  // std::cout << "xmin: " << xmin << std::endl;
  // std::cout << "ymin: " << ymin << std::endl;
  // std::cout << "zmin: " << zmin << std::endl;
  
  double minEnergyCoorDouble[nAtom*3];
  for(int i = 0; i < nAtom; i++)
  {
    minEnergyCoorDouble[i*3 + 0] = (double) minEnergyCoor[i*3 + 0];
    minEnergyCoorDouble[i*3 + 1] = (double) minEnergyCoor[i*3 + 1];
    minEnergyCoorDouble[i*3 + 2] = (double) minEnergyCoor[i*3 + 2];
  }

  for(int i = 0; i < nAtom; i++)
  {
    minEnergyCoorDouble[i*3 + 0] += (xmin - minEnergyMinX + minEnergyIdxX * spacing);
    minEnergyCoorDouble[i*3 + 1] += (ymin - minEnergyMinY + minEnergyIdxY * spacing);
    minEnergyCoorDouble[i*3 + 2] += (zmin - minEnergyMinZ + minEnergyIdxZ * spacing);
  }

  mol.SetCoordinates(minEnergyCoorDouble);
  fileName = "TranRotaSearch";
  fileName += ".pdb";
  conv.WriteFile(&mol, fileName);

  std::cout << "MinEnergy: " << minEnergy << std::endl;
  return 0;
}

#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <vector>
#include <string>
#include <math.h>
#include <time.h>

#include "OpenMM.h"
#include <openbabel/obconversion.h>
#include <openbabel/mol.h>
#include <hipfft/hipfft.h>
#include "ReadCrd.h"
#include "ReadGrids.h"
#include "ReadQuaternions.h"
#include "Rotate.h"
#include "GetNonbondedParameters.h"
#include "GetMinCoors.h"
#include "GetMaxCoors.h"
#include "GetIdxOfAtomsForVdwRadius.h"
#include "FillLigandGrid.h"
#include "GeneConformations.h"
#include "kernel.h"
#include "QuaternionUniformSampling.h"

#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 

//// main function ////
int main(int argc, char** argv)
{
  OpenMM::Platform::loadPluginsFromDirectory(
  					     "/home/xqding/apps/openmmDev/lib/plugins");
  
  // read molecule
  std::string fileName(argv[1]);
  OpenBabel::OBMol mol;
  OpenBabel::OBConversion conv(&std::cin, &std::cout);
  conv.SetInFormat("mol2");
  conv.SetOutFormat("pdb");
  conv.ReadFile(&mol, fileName);
  int nAtom = mol.NumAtoms();
  
  // read system
  std::ifstream sysFile;
  sysFile.open(argv[2], std::ifstream::in);
  if (sysFile.fail())
  {
    std::cout << "Open system file failed: " << argv[4] << std::endl;    return 1;
  }
  OpenMM::System *sys = new OpenMM::System();
  sys = OpenMM::XmlSerializer::deserialize<OpenMM::System>(sysFile);
  
  // get nonbonded parameters
  float atomCharges[nAtom];
  float atomEpsilons[nAtom];
  float atomRadii[nAtom];
  GetNonbondedParameters(sys, atomCharges, atomEpsilons, atomRadii);

  // read grid potential
  int numOfGrids, xdim, ydim, zdim;
  float midx, midy, midz;
  float xlen, ylen, zlen;
  float spacing, restraint_k;
  float *gridRadii, *gridValues;
  ReadGrids(numOfGrids, xdim, ydim, zdim,
  	    midx, midy, midz,
  	    xlen, ylen, zlen,
  	    spacing, restraint_k,
  	    gridRadii, gridValues,
  	    argv[3]);
  int numOfVdwGrids = numOfGrids - 1;
  
  // add electrostatic grid force
  double xmin = midx - xlen / 2;
  double ymin = midy - ylen / 2;
  double zmin = midz - zlen / 2;
  double xmax = xmin + (xdim - 1) * spacing;
  double ymax = ymin + (ydim - 1) * spacing;
  double zmax = zmin + (zdim - 1) * spacing;

  // get index of atoms for each vdw radius
  int numOfVdwGridsUsed;
  std::vector<int> idxOfVdwUsed;
  std::vector< std::vector<int> > idxOfAtomVdwRadius(numOfVdwGrids);
  GetIdxOfAtomsForVdwRadius(nAtom, atomRadii,
  			    numOfVdwGrids, gridRadii,
  			    numOfVdwGridsUsed, idxOfVdwUsed,
  			    idxOfAtomVdwRadius);
  int numOfGridsUsed = numOfVdwGridsUsed + 1;

  // copy out the potential grids which are used
  float *usedGridValues;
  usedGridValues = new float[numOfGridsUsed*xdim*ydim*zdim];
  for(int i = 0; i < numOfVdwGridsUsed; i++)
  {
    memcpy(&usedGridValues[i*xdim*ydim*zdim],
  	   &gridValues[idxOfVdwUsed[i]*xdim*ydim*zdim],
  	   sizeof(float)*xdim*ydim*zdim);
  }
  memcpy(&usedGridValues[numOfVdwGridsUsed*xdim*ydim*zdim],
  	 &gridValues[numOfVdwGrids*xdim*ydim*zdim],
  	 sizeof(float)*xdim*ydim*zdim);

  //// do translation and rotation search using FFT
  // batch cudaFFT for potential grids
  int n[3];
  n[0] = xdim;
  n[1] = ydim;
  n[2] = zdim;
  int inembed[3];
  inembed[0] = xdim;
  inembed[1] = ydim;
  inembed[2] = zdim;
  int idist = inembed[0] * inembed[1] * inembed[2];
  int istride = 1;
  
  int onembed[3];
  onembed[0] = xdim;
  onembed[1] = ydim;
  onembed[2] = zdim/2 + 1;
  int odist = onembed[0] * onembed[1] * onembed[2];
  int ostride = 1;
  int nBatchPotential = numOfGridsUsed;
  
  hipfftReal* d_potential_f;
  hipMalloc((void **)&d_potential_f, sizeof(hipfftReal)*nBatchPotential*idist);
  hipMemcpy(d_potential_f, usedGridValues,
  	     sizeof(hipfftReal)*nBatchPotential*idist,
  	     hipMemcpyHostToDevice);
  hipfftComplex *d_potential_F;
  hipMalloc((void **)&d_potential_F, sizeof(hipfftComplex)*nBatchPotential*odist);
  hipfftHandle potentialPlan;
  hipfftResult potentialRes = hipfftPlanMany(&potentialPlan, 3, n,
  					   inembed, istride, idist,
  					   onembed, ostride, odist,
  					   HIPFFT_R2C, nBatchPotential);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "Potential plan creat failed!";
    return 1;
  }
  potentialRes = hipfftExecR2C(potentialPlan, d_potential_f, d_potential_F);
  if (potentialRes != HIPFFT_SUCCESS)
  {
    std::cout << "Potential transform failed!";
    return 1;
  }

  // // read quaternions
  // int numOfTotalQuaternions = atoi(argv[4]);
  // std::cout << "numOfTotalQuaternions: " << numOfTotalQuaternions << std::endl;
  // float *quaternions;
  // quaternions = new float[numOfTotalQuaternions * 4];
  // ReadQuaternions(numOfTotalQuaternions, quaternions, argv[5]);

  // generate uniform quaternions
  int numOfTotalQuaternions = atoi(argv[4]);
  float* quaternions;
  std::random_device rd;
  std::mt19937_64 gen(rd());
  QuaternionUniformSampling(gen, quaternions, numOfTotalQuaternions);

  // get coor
  float* coor;
  coor = new float[nAtom*3];
  for(int i = 0; i < nAtom; i++)
  {
    coor[i*3 + 0] = (float) mol.GetCoordinates()[i*3 + 0];
    coor[i*3 + 1] = (float) mol.GetCoordinates()[i*3 + 1];
    coor[i*3 + 2] = (float) mol.GetCoordinates()[i*3 + 2];
  }

  // ignore quaterions, whose end structures' dimenstion is larger than the grids
  // rotate
  float* coors_all_quaters;
  coors_all_quaters = new float[numOfTotalQuaternions*nAtom*3];
  for(int i = 0; i < numOfTotalQuaternions; i++)
  {
    for(int j = 0; j < nAtom; j++)
    {
      Rotate(&quaternions[i*4], &coor[j*3], &coors_all_quaters[i*nAtom*3+j*3]);
    }
  }

  float mincoors_all[numOfTotalQuaternions*3]; // minimium coordinates along x, y, and z for all quaternions
  float maxcoors_all[numOfTotalQuaternions*3]; // maximum coordinates along x, y, and z for all quaternions
  float ligandLength_all[numOfTotalQuaternions*3]; // lenth along x, y and z for each orientation for all quaternions

  // calculate minimum coor for each quaternions
  GetMinCoors(numOfTotalQuaternions, nAtom, coors_all_quaters, mincoors_all);

  // calculate maximum coor for each quaternions
  GetMaxCoors(numOfTotalQuaternions, nAtom, coors_all_quaters, mincoors_all);

  // calculate the length for each quaternion
  for(int i = 0; i < numOfTotalQuaternions; i++)
  {
    ligandLength_all[i*3 + 0] = maxcoors_all[i*3 + 0] - mincoors_all[i*3 + 0];
    ligandLength_all[i*3 + 1] = maxcoors_all[i*3 + 1] - mincoors_all[i*3 + 1];
    ligandLength_all[i*3 + 2] = maxcoors_all[i*3 + 2] - mincoors_all[i*3 + 2];
  }

  // index of quaternions which keep the ligand dimenstion smaller than grids
  std::vector <int> idxOfQuatersUsed;
  for(int i = 0; i < numOfTotalQuaternions; i++)
  {
    if(ligandLength_all[i*3 + 0] < xdim && ligandLength_all[i*3 + 1] < ydim && ligandLength_all[i*3 + 2] < zdim)
    {
      idxOfQuatersUsed.push_back(i);
    }
  }

  size_t numOfQuaternionsUsed = idxOfQuatersUsed.size();
  std::cout << "numOfQuaternionsUsed: " << numOfQuaternionsUsed << std::endl;
  float* quaternionsUsed;
  quaternionsUsed = new float[numOfQuaternionsUsed*4];
  for(int i = 0; i < numOfQuaternionsUsed; i++)
  {
    quaternionsUsed[i*4 + 0] = quaternions[idxOfQuatersUsed[i]*4 + 0];
    quaternionsUsed[i*4 + 1] = quaternions[idxOfQuatersUsed[i]*4 + 1];
    quaternionsUsed[i*4 + 2] = quaternions[idxOfQuatersUsed[i]*4 + 2];
    quaternionsUsed[i*4 + 3] = quaternions[idxOfQuatersUsed[i]*4 + 3];
  }
  
  // loop over all batches for different orientation
  int numOfQuaternionsOneBatch = 50;
  int numOfBatches = numOfQuaternionsUsed / numOfQuaternionsOneBatch + 1;
    
  // allocate the data structures which will be used
  float *coors; // rotated coordinates
  coors = new float[numOfQuaternionsOneBatch*nAtom*3];
  float mincoors[numOfQuaternionsOneBatch*3]; // minimium coordinates along x, y, and z
  float maxcoors[numOfQuaternionsOneBatch*3]; // maximum coordinates along x, y, and z
  float ligandLength[numOfQuaternionsOneBatch*3]; // lenth along x, y and z for each orientation
  
  float *ligandGridValues; // grid for ligand
  ligandGridValues = new float[numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim];
  
  // cudaFFT for ligand grid
  int nBatchLigand = numOfQuaternionsOneBatch*numOfGridsUsed;
  hipfftReal* d_ligand_f;
  hipMalloc((void **)&d_ligand_f, sizeof(hipfftReal)*nBatchLigand*idist);
  hipfftComplex * d_ligand_F;
  hipMalloc((void **)&d_ligand_F, sizeof(hipfftComplex)*nBatchLigand*odist);
  hipfftHandle ligandPlan;
  hipfftResult ligandRes = hipfftPlanMany(&ligandPlan, 3, n,
  					inembed, istride, idist,
  					onembed, ostride, odist,
  					HIPFFT_R2C, nBatchLigand);
  if (ligandRes != HIPFFT_SUCCESS)
  {
    std::cout << "ligand plan creat failed!";
    return 1;
  }

  dim3 threads_ConjMult(1024, 1, 1);
  dim3 blocks_ConjMult((numOfQuaternionsOneBatch*numOfGridsUsed*odist)/(1024*1024) + 1,1024,1);
 
  hipfftComplex * d_ligand_sum_F;
  hipMalloc((void **)&d_ligand_sum_F, sizeof(hipfftComplex)*numOfQuaternionsOneBatch*odist);
  
  dim3 threads_SumGrids(1024, 1, 1);
  dim3 blocks_SumGrids((numOfQuaternionsOneBatch*odist)/(1024*1024) + 1,1024,1);
  
  hipfftReal *d_ligand_sum_f;
  hipMalloc((void **)&d_ligand_sum_f, sizeof(hipfftReal)*numOfQuaternionsOneBatch*idist);
  hipfftHandle ligandRPlan;
  hipfftResult ligandRRes = hipfftPlanMany(&ligandRPlan, 3, n,
  					 onembed, ostride, odist,
  					 inembed, istride, idist,
  					 HIPFFT_C2R, numOfQuaternionsOneBatch);
  if (ligandRRes != HIPFFT_SUCCESS)
  {
    std::cout << "ligand reverse plan creat failed!" << std::endl;
    std::cout << "Error code: " << ligandRRes << std::endl;
    return 1;
  }
  float* energy;
  energy = new float[numOfQuaternionsOneBatch*idist];

  int minEnergyQ = 0;
  int minEnergyIdxX = 0;
  int minEnergyIdxY = 0;
  int minEnergyIdxZ = 0;
  float minEnergy = INFINITY;
    
  for(int idxOfBatch = 0; idxOfBatch < numOfBatches; idxOfBatch++)
  {
    std::cout << "idxOfBatch: " << idxOfBatch << std::endl;
    // rotate
    for(int i = 0; i < numOfQuaternionsOneBatch; i++)
    {
      if (idxOfBatch*numOfQuaternionsOneBatch + i < numOfQuaternionsUsed)
      {
	for(int j = 0; j < nAtom; j++)
	{
	  Rotate(&quaternionsUsed[(idxOfBatch*numOfQuaternionsOneBatch + i)*4], &coor[j*3], &coors[i*nAtom*3+j*3]);
	}
      }
    }
    
    // calculate minimum coor for each quaternions
    GetMinCoors(numOfQuaternionsOneBatch, nAtom, coors, mincoors);

    // calculate maximum coor for each quaternions
    GetMaxCoors(numOfQuaternionsOneBatch, nAtom, coors, maxcoors);

    // calculate the length for each quaternion
    for(int i = 0; i < numOfQuaternionsOneBatch; i++)
    {
      ligandLength[i*3 + 0] = maxcoors[i*3 + 0] - mincoors[i*3 + 0];
      ligandLength[i*3 + 1] = maxcoors[i*3 + 1] - mincoors[i*3 + 1];
      ligandLength[i*3 + 2] = maxcoors[i*3 + 2] - mincoors[i*3 + 2];
    }
      
    // fill ligand grid
    memset(ligandGridValues, 0, sizeof(float)*numOfQuaternionsOneBatch*numOfGridsUsed*xdim*ydim*zdim);
    FillLigandGrid(numOfQuaternionsOneBatch,
		   nAtom, coors, mincoors,
		   atomCharges, atomEpsilons,
		   numOfVdwGridsUsed, idxOfVdwUsed,
		   idxOfAtomVdwRadius,
		   xdim, ydim, zdim,
		   spacing, ligandGridValues);


    // batch cudaFFT for ligand grid
    hipMemcpy(d_ligand_f, ligandGridValues,
	       sizeof(hipfftReal)*nBatchLigand*idist,
	       hipMemcpyHostToDevice);
    ligandRes = hipfftExecR2C(ligandPlan, d_ligand_f, d_ligand_F);
    if (ligandRes != HIPFFT_SUCCESS)
    {
      std::cout << "ligand transform failed!";
      return 1;
    }

    // calcualte energy using reverse FFT
    ConjMult <<<blocks_ConjMult, threads_ConjMult>>> (d_potential_F, d_ligand_F, odist, numOfGridsUsed);
    CUDA_CHECK();

    SumGrids <<<blocks_SumGrids, threads_SumGrids>>> (d_ligand_F, d_ligand_sum_F, numOfGridsUsed, odist, idist);
    CUDA_CHECK();

    ligandRRes = hipfftExecC2R(ligandRPlan, d_ligand_sum_F, d_ligand_sum_f);
    if (ligandRRes != HIPFFT_SUCCESS)
    {
      std::cout << "ligand reverse transform failed!";
      return 1;
    }

    // copy energy back
    hipMemcpy(energy, d_ligand_sum_f, sizeof(float)*numOfQuaternionsOneBatch*idist,
	       hipMemcpyDeviceToHost);

    // record the minimum energy pose in terms of quaternions, x, y and z
    for(int q = 0; q < numOfQuaternionsOneBatch; q++)
    {
      for(int i = 0; i < (xdim-int(ligandLength[q*3+0]/spacing)-2); i++)
      {
	for(int j = 0; j < (ydim-int(ligandLength[q*3+1]/spacing)-2); j++)
	{
	  for(int k = 0; k < (zdim-int(ligandLength[q*3+2]/spacing)-2); k++)
	  {
	    if(idxOfBatch*numOfQuaternionsOneBatch + q < numOfQuaternionsUsed)
	    {
	      int tmp = q*idist + (i*ydim + j)*zdim + k;
	      if((energy[tmp]/sqrt(idist)) < minEnergy)
	      {
		minEnergy = energy[tmp] / sqrt(idist);
		minEnergyQ = idxOfBatch * numOfQuaternionsOneBatch + q;
		minEnergyIdxX = i;
		minEnergyIdxY = j;
		minEnergyIdxZ = k;
	      }
	    }
	  }
	}
      }
    }
  }

  // calculate the coordinates corresponding to minimum energy
  float minEnergyCoor[nAtom*3];
  for(int i = 0; i < nAtom; i++)
  {
    Rotate(&quaternionsUsed[minEnergyQ*4], &coor[i*3], &minEnergyCoor[i*3]);
  }
  
  float minEnergyMinX = minEnergyCoor[0];
  float minEnergyMinY = minEnergyCoor[1];
  float minEnergyMinZ = minEnergyCoor[2];
  for(int i = 1; i < nAtom; i++)
  {
    if (minEnergyCoor[i*3+0] < minEnergyMinX) { minEnergyMinX = minEnergyCoor[i*3+0]; }
    if (minEnergyCoor[i*3+1] < minEnergyMinY) { minEnergyMinY = minEnergyCoor[i*3+1]; }
    if (minEnergyCoor[i*3+2] < minEnergyMinZ) { minEnergyMinZ = minEnergyCoor[i*3+2]; }
  }
  
  double minEnergyCoorDouble[nAtom*3];
  for(int i = 0; i < nAtom; i++)
  {
    minEnergyCoorDouble[i*3 + 0] = (double) minEnergyCoor[i*3 + 0];
    minEnergyCoorDouble[i*3 + 1] = (double) minEnergyCoor[i*3 + 1];
    minEnergyCoorDouble[i*3 + 2] = (double) minEnergyCoor[i*3 + 2];
  }

  for(int i = 0; i < nAtom; i++)
  {
    minEnergyCoorDouble[i*3 + 0] += (xmin - minEnergyMinX + minEnergyIdxX * spacing);
    minEnergyCoorDouble[i*3 + 1] += (ymin - minEnergyMinY + minEnergyIdxY * spacing);
    minEnergyCoorDouble[i*3 + 2] += (zmin - minEnergyMinZ + minEnergyIdxZ * spacing);
  }

  mol.SetCoordinates(minEnergyCoorDouble);
  fileName = "TranRotaSearch";
  fileName += ".pdb";
  conv.WriteFile(&mol, fileName);

  std::cout << "MinEnergy: " << minEnergy << std::endl;
  return 0;
}
